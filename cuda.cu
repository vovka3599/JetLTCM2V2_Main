#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_f.h"

__global__ void PrintId()
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  printf("Cuda thread id = %d\n", idx);
}

extern "C" void test()
{
    dim3 blocks(4,1,1);
    dim3 thread(1024,1,1);

    hipEvent_t start, stop;
    float gpuTime;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    PrintId<<<blocks, thread>>>();
    hipDeviceSynchronize();

    hipEventRecord( stop, 0 );
    hipEventSynchronize(stop);
    hipEventElapsedTime( &gpuTime, start, stop );
    printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);
    hipEventDestroy( start);
    hipEventDestroy( stop);
}